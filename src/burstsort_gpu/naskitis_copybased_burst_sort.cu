#include "hip/hip_runtime.h"
/*******************************************************************************
 * // Begin statement                                                          *
 *                                                                             *
 * Author:        Dr. Nikolas Askitis                                          *
 * Email:         askitisn@gmail.com                                           *
 * Github.com:    https://github.com/naskitis                                  *
 *                                                                             *
 * Copyright @ 2016.  All rights reserved.                                     *
 *                                                                             *
 * Permission to use my software is granted provided that this statement       *
 * is retained.                                                                *
 *                                                                             *
 * My software is for non-commercial use only.                                 *
 *                                                                             *
 * If you want to share my software with others, please do so by               *
 * sharing a link to my repository on github.com.                              *
 *                                                                             *
 * If you would like to use any part of my software in a commercial or public  *
 * environment/product/service, please contact me first so that I may          *
 * give you written permission.                                                *
 *                                                                             *
 * This program is distributed without any warranty; without even the          *
 * implied warranty of merchantability or fitness for a particular purpose.    *
 *                                                                             *
 * Note:                                                                       *
 *                                                                             *
 * The copy-based burst sort algorithm is one of the fastest in-memory         *
 * sorting algorithms for variable-length strings, as detailed in the          *
 * the computing literature. Nonetheless, the original implementation of       *
 * the copy-based burst sort by its authors is not readily available. As       *
 * such, I have written my own implementation of the copy-based burst sort     *
 * in C, using my knowledge of cache-conscious string data structures. I am    *
 * not sure if my implementation is as fast or as memory-efficient as the      *
 * original used by the authors, but I am confident that it can serve as       *
 * a good baseline for your experiments :)                                     *
 *                                                                             *
 * My implementation of the copy-based burst sort algorithm is a little        *
 * different from how its explained in literature. First, the strings that     *
 * are stored in containers are length-encoded --- the original algorithm      *
 * simply stored null-terminated strings. Second, containers are not sized     *
 * to fit into the L2 cache of a typical CPU, as is the intention of the       *
 * burst sort algorithm. Instead, much like my implementation of the array     *
 * burst trie and HAT-trie, containers are burst once they store more than a   *
 * given number of strings.                                                    *
 *                                                                             *
 * (Usage: ./naskitis_copybased-burst_sort                                     *
 *                                [container-size] [number-of-files-to-insert] *
 *                                [file1] [file2] ... )                        *
 * Output: (printed to stderr)                                                 *
 * Copybased burst sort 520.94 446.67 12.60 28772169 64 ...                    *
 * [algo]               [virtual mem] [estimated mem] [time to build]          *
 *                      [num keys inserted] [container size] ...               *
 * // End statement                                                            *
 ******************************************************************************/
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "include/common.h"
#include "sort_module.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <assert.h>

#define BUCKET_OVERHEAD (2 + sizeof(uint32_t))
#define STRING_EXHAUST_TRIE 31
#define STRING_EXHAUST_CONTAINER 2
#define CONSUMED 0
#define ALLOC_OVERHEAD 16

/* array of pointers used to sort a bucket */
ptr_struct *str_ptr;

/* stores the path of characters encountered as you traverse a trie */
char *path;

/* variables needed to maintain trie nodes */
char **trie_pack=NULL;
__device__ char **trie_pack_device=NULL;
uint32_t trie_pack_idx=0;
__device__ uint32_t trie_pack_idx_device=0;
uint32_t trie_counter=0;
__device__ uint32_t trie_counter_device=0;
uint32_t trie_pack_entry_capacity=32768;
__device__ uint32_t trie_pack_entry_capacity_device=32768;
uint32_t trie_pack_capacity=256;
__device__ uint32_t trie_pack_capacity_device=256;
uint32_t total_trie_pack_memory=0;
char *trie_buffer;
char *current_bucket;
char *root_trie;
__device__ char* root_trie_device;
int increment=0;
int copy_increment=0;
uint64_t BUCKET_SIZE_LIM=35;
uint64_t inserted=0;
uint64_t searched=0;
uint64_t depth=0;
uint64_t num_buckets=0;
uint64_t num_tries=0;
uint64_t trie_buffer_capacity = 65536;
__device__ uint64_t trie_buffer_capacity_device = 65536;
uint64_t trie_buffer_size = 0;
uint64_t bucket_mem=0;
uint64_t max_trie_depth=0;
uint64_t depth_accumulator=0;
uint64_t mtf_counter=0;

void  destroy();
void split_container(char *, char **);
//__global__
void burst_container(char *, char, char **);
void resize_container(char **, uint32_t, uint32_t);

	
uint32_t add_to_bucket_no_search(char *bucket,  
                     char path, 
		     char *query_start, 
		     char **c_trie);
		     
uint32_t add_to_bucket_no_search_with_len(char *bucket,  
                     char path, 
		     char *query_start, 
		     char **c_trie, int len);
const int N = 16; 
const int blocksize = 16; 
char *dev_string1;
char *dev_string2;
char *array_dev;
char *query_start_dev;
char **bucket_device;
char *word_device;
hipStream_t mystream1;
hipStream_t mystream2;
hipStream_t mystream3;
__global__ 
void hello(char *a, int *b) 
{
	printf("%d",threadIdx.x);
	a[threadIdx.x] += b[threadIdx.x];

}
__global__
void copy_string(char* dev_stri,char* dev_stri2,int len)
{




  int tID=blockIdx.x*blockDim.x+threadIdx.x;
  int i=0;
 if(*(dev_stri2+tID)!='\0')
   {
        *(dev_stri+tID)=*(dev_stri2+tID);
        len--;

   }
   *(dev_stri+tID+1)='\0';
  __syncthreads();


}

__device__
int new_container_shared(char **c_trie_dev, char path, char *word)
{
  char x[2 + sizeof(uint32_t)];

  /* allocate space for the container */

  
  //printf("in device container");
 // if (x==NULL) fatal (MEMORY_EXHAUSTED);

  
  *x=0;
  *(uint32_t *)(x+31)=0;

  //*(c_trie_dev +path)=x;

 if( *word == '\0')
  {
    *(uint32_t *)(x+31)=1;
  }
  else
  {
   // add_to_bucket_no_search(x, path, word, c_trie);
   //printf("add to bucket no search");
    char *array_start, *query;
    char *array;
  char *tmp=*(c_trie_dev+path);

  char *consumed=0;
  uint32_t array_offset;
  uint32_t register len;
  uint32_t num=0;

  array = (char *)(x+31);
  consumed = (char *)(x+2);

  array_start=array;
  query = word;
  if(*consumed == 0) { *consumed = 1; goto insert; }
  insert:
   while( *array != '\0')
  {

    if( ( len = (unsigned int) *array ) >= 128 )
    {
      len = (unsigned int) ( ( *array & 0x7f ) << 8 ) |  (unsigned int) ( *(++array) & 0xff );
    }
    array = (array+1) + len;
    ++num;
  }
   //len=slen(query);
   for(; *query != '\0'; query++);//changed

    len = query - word;//changed


  
  array_offset = array-array_start;

  
 // resize_container((char **)(c_trie+path), array_offset, ( len < 128 ) ? len+2 : len+3);

 
  array = (char *)( *(c_trie_dev+path) + 2);
  array_start=array;
  array += array_offset; 
  //printf("in add_to_searc_no_len");
  char *ab="asdf";
/* int tid=threadIdx.x;
if(*(word+tid)!='\0')
  {
        *word++;
        printf("%d",tid);
        len--;
  }*/
//  if (threadIdx.x == 0){
  copy_string<<<1,len>>>(ab,word,len);
//}
  // *array='\0';
 int tID=blockIdx.x*blockDim.x+threadIdx.x;
 int i=0;
 if(*(word+tID)!='\0')
   {
        //*(array+tID)=*(w+tID);
        len--;
     //  printf("in device function");

   }
   *(word+tID+1)='\0';


  //increment++;


  }
  return 1;
}
__device__
uint32_t add_to_bucket_no_search_shared(char *x, char *path,char *word, char **c_trie)
{
  char *array, *array_start, *query;
  //char *tmp=*(c_trie+path);

  char *consumed=0;
  uint32_t array_offset;
  uint32_t register len;
  uint32_t num=0;

  array = (char *)(x+31);
  consumed = (char *)(x+2);

  array_start=array;
  query = word;

  
  if(*consumed == 0) { *consumed = 1; goto insert; }

  
  while( *array != '\0')
  {

    if( ( len = (unsigned int) *array ) >= 128 )
    {
      len = (unsigned int) ( ( *array & 0x7f ) << 8 ) |  (unsigned int) ( *(++array) & 0xff );
    }
    array = (array+1) + len;
    ++num;
  }

  insert:

 
  for(; *query != '\0'; query++);//changed

  len = query - word;//changed

   //len=slen(query);

  
  array_offset = array-array_start;

  
  //resize_container((char **)(c_trie+path), array_offset, ( len < 128 ) ? len+2 : len+3);

 
  array = (char *)( *(c_trie+*path) + BUCKET_OVERHEAD);
  array_start=array;
  array += array_offset;
  //increment++;
  
 // if(increment<1500){

  if( len < 128 )
  {
      *array = (char) len;

  }
  
  else
  {
    *array     = (char) ( len >> 8) | 0x80;
    *(++array) = (char) ( len ) & 0xff;

  }
  array++;
  printf("in the add to bucket search no len");
/* thrust::host_vector<char> G(sizeof(array));
  for(int i=0;i<sizeof(array);i++)
  {
        G[i]=array[i];
  }
  thrust::device_vector<char> C = G;*/

  /* copy the string into the array */
 /* while( *query_start != '\0')
  {
    *array++ = *query_start++;
  }*/
 // hipProfilerStart();
 /* hipStream_t mystream1;
   hipStream_t mystream2;
   hipStream_t mystream3;
  hipStreamCreate(&mystream1);
  hipStreamCreate(&mystream2);
  hipStreamCreate(&mystream3);*/
 /* hipStreamCreateWithFlags(&mystream1,hipStreamNonBlocking);
   hipStreamCreateWithFlags(&mystream2,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&mystream3,hipStreamNonBlocking);*/

 /* hipMemcpyAsync(&array_dev[0],&array[0],300*sizeof(char), hipMemcpyHostToDevice);
  hipMemcpyAsync(&query_start_dev[0],&query_start[0],300*sizeof(char), hipMemcpyHostToDevice);
  copy_string_to_array<<<1,len,0,mystream1>>>(&array_dev[0],&query_start_dev[0],len);
  //hipDeviceSynchronize();
  hipMemcpyAsync(&array[0],&array_dev[0],300*sizeof(char), hipMemcpyDeviceToHost);*/
 /*  while( *query_start != '\0')
  {
    *array++ = *query_start++;
  }
   *array='\0';
}
++num;*/
return 1;
}


/* resize a container, using the techniques I developed for the array hash table */
void resize_container(char **bucket, uint32_t array_offset, uint32_t required_increase)
{
    #ifdef EXACT_FIT

    char *tmp = malloc(array_offset + required_increase + BUCKET_OVERHEAD );
    if(tmp == NULL) fatal (MEMORY_EXHAUSTED);

    /* copy the existing array into the new one */
    if(array_offset==0)  
    {  
      memcpy(tmp, *bucket, array_offset+BUCKET_OVERHEAD);
    } 
    else
    {
      /* once extra byte to transfer the end-of-container flag */
      memcpy(tmp, *bucket, array_offset+1+BUCKET_OVERHEAD);  
    }

    /* free the old array and assign the container pointer to the new array */ 
    free( *bucket );
    *bucket = tmp;
 
    /* else grow the array in blocks or pages */
    #else 

    if(array_offset==0)
    {
      /* otherwise, grow the array with paging */
      /* if the required space is less than 32 bytes, than allocate a 32 byte block */
      if(required_increase + BUCKET_OVERHEAD <= _32_BYTES)
      {
        char *tmp =(char*) malloc(_32_BYTES);
        if(tmp == NULL) fatal (MEMORY_EXHAUSTED);

        memcpy(tmp, *bucket, array_offset+BUCKET_OVERHEAD);

        /* free the old array and assign the container pointer to the new array */ 
        free( *bucket );
        *bucket = tmp; 
      }
      /* otherwise, allocate as many 64-byte blocks as required */
      else
      {
        uint32_t number_of_blocks = ((int)( (required_increase - 1 + BUCKET_OVERHEAD) >> 6)+1);   

        char *tmp =(char*) malloc(number_of_blocks << 6);
        if(tmp == NULL) fatal (MEMORY_EXHAUSTED);

        memcpy(tmp, *bucket, array_offset+BUCKET_OVERHEAD);

        /* free the old array and assign the container pointer to the new array */ 
        free( *bucket );
        *bucket = tmp; 
      }

      return;
    }

    uint32_t old_array_size = array_offset + 1 + BUCKET_OVERHEAD;
    uint32_t new_array_size = (array_offset + required_increase + BUCKET_OVERHEAD);
    
    /* if the new array size can fit within the previously allocated 32-byte block, 
     * then no memory needs to be allocated.
     */
    if ( old_array_size <= _32_BYTES  &&  new_array_size <= _32_BYTES )
    {
      return;
    }
    /* if the new array size can fit within a 64-byte block, then allocate only a
     * single 64-byte block.
     */
    else if ( old_array_size <= _32_BYTES  &&  new_array_size <= _64_BYTES)
    {  
      char *tmp =(char*) malloc(_64_BYTES);
      if(tmp == NULL) fatal (MEMORY_EXHAUSTED);
      
      /* copy the old array into the new */
      memcpy(tmp, *bucket, old_array_size);
      
      /* delete the old array */ 
      free( *bucket );

      /* assign the container pointer to the new array */
      *bucket = tmp;

      return;
    }
    /* if the new array size can fit within a 64-byte block, then return */
    else if  (old_array_size <= _64_BYTES && new_array_size <= _64_BYTES )
    {
      return;
    }
    /* resize the current array by as many 64-byte blocks as required */
    else
    {
      uint32_t number_of_blocks = ((int)( (old_array_size-1) >> 6) + 1);
      uint32_t number_of_new_blocks = ((int)( (new_array_size-1) >> 6) + 1);

      if(number_of_new_blocks > number_of_blocks)
      {
        /* allocate as many blocks as required */
        char *tmp =(char*) malloc(number_of_new_blocks << 6);
        if (tmp==NULL) fatal(MEMORY_EXHAUSTED);
        //hipMemcpyAsync(dev_string1,tmp,200*sizeof(char), hipMemcpyHostToDevice);
      /*  for (int i = 0; i < 20; i++) {
                hipMalloc((void**)&bucket_device[i], 20* sizeof(char*));
                hipMemcpy(bucket_device[i], bucket[i], 20*sizeof(char*),hipMemcpyHostToDevice);
             }*/
       // hipMemcpyAsync(bucket_device,bucket,200*sizeof(char*), hipMemcpyHostToDevice);
       // hipMemcpyAsync(array,dev_string1,200*sizeof(char), hipMemcpyDeviceToHost);
        /* copy the old array, a word at a time, into a new array */
        node_cpy( (uint32_t *) tmp, (uint32_t *) *bucket, number_of_blocks<<6); 
      //  hipMemcpyAsync(tmp,dev_string1,200*sizeof(char), hipMemcpyHostToDevice);
        
         /* for (int i = 0; i < 20; i++) {
              //  hipMalloc((void**)&bucket_device[i], 200* sizeof(char*));
                hipMemcpy(bucket[i], bucket_device[i], 20*sizeof(char*),hipMemcpyDeviceToHost);
             }*/
        /* free the old array */
        free( *bucket );
        
        /* assign the container pointer to the new array */
        *bucket = tmp;
      } 
    } 

  #endif 
}	     
    
/* need to implement if it runs of out packs. See source of HAT-trie for more details. */
char * new_trie()
{
  if(trie_counter == trie_pack_entry_capacity)
  {
    trie_pack_idx++;
    assert(trie_pack_idx<128);

    *(trie_pack+trie_pack_idx) =(char*) calloc(trie_pack_entry_capacity*TRIE_SIZE, sizeof(char));
    trie_counter=0;
  }

  return *(trie_pack + trie_pack_idx) + (trie_counter++ * TRIE_SIZE);
}


/* take a pointer and return 1 if it points to a trie node.  This can
 * be determined by checking whether the address lies within the blocks
 * of memory used to store the trie nodes 
 */
int is_it_a_trie(char *x)
{
  register int idx=0;
  for(; idx <= trie_pack_idx; idx++)
  { 
     if ( x >= *(trie_pack+idx) && x <= (*(trie_pack+idx)+(TRIE_SIZE * (trie_pack_entry_capacity-1) )) ) 
       return 1;
  } 

  return 0;
}
__device__ int is_it_a_trie_shared(char *x_shared,uint32_t trie_pack_entry_capacity_shared,uint32_t trie_pack_idx_shared,char **trie_pack_shared)
{
  register int idx=0;
  for(; idx <= trie_pack_idx_shared; idx++)
  {
     if (( x_shared >= *(trie_pack_shared)+idx) && x_shared <= (*(trie_pack_shared+idx)+(1024 * (trie_pack_entry_capacity_shared-1) ))) 
       
       return 1;
  }
  

  return 0;
}


/*__device__ char* new_trie_device()
{
  if(trie_counter_device == trie_pack_entry_capacity_shared)
  {
    trie_pack_idx_shared++;
    assert(trie_pack_idx_shared<128);

    //*(trie_pack+trie_pack_idx) =(char*) calloc(trie_pack_entry_capacity*TRIE_SIZE, sizeof(char));
    trie_counter_shared=0;
  }

  return *(trie_pack_shared + trie_pack_idx_shared) + (trie_counter_shared++ * TRIE_SIZE_shared);
}*/

/* initialize the burst trie structure */
//__global__
__global__ void printing()
{
	printf("naskitis fjoiwjefiowej");
}
__global__ void insert_device(char *word)
{
  

  char **c_trie_shared=NULL;
//  char **trie_pack_shared=NULL;
  //int i=0;

 
  //trie_pack_device = (char **) calloc (trie_pack_capacity_device, sizeof(char *));
  uint32_t trie_pack_idx_shared=0;
  uint32_t trie_counter_shared=0;
  uint32_t trie_pack_entry_capacity_shared=34234;
 // trie_pack = (char **) calloc (trie_pack_capacity, sizeof(char *));
 // char* ptr = (char*)malloc(123);
 //char ptr[123];
 char *trie_pack_shared[34234];
// *trie_pack_shared=*(trie_pack_shared+trie_pack_idx_shared);
//char *trie_pack_shared[1024];
  
//* trie_pack_shared =(char*) malloc((1024));
  if(trie_counter_shared == trie_pack_entry_capacity_shared)
  {
    trie_pack_idx_shared++;
   // assert(trie_pack_idx_shared<128);
   * trie_pack_shared=*(trie_pack_shared+trie_pack_idx_shared);

    *trie_pack_shared =(char*) malloc(32324*1024);
    trie_counter_shared=0;
  }

  
char*  root_trie_shared[1024*126];
 c_trie_shared = (char**)root_trie_shared;

  
 for(int i=0; i<128; i++) {
     *(c_trie_shared+i)=NULL;
  }

  
  *(c_trie_shared+31)=0;
  
 
  char *x_shared;
  int r_shared=0;


 // char *word_device;
  //hipMalloc((void **)&word_device, sizeof(word));
 // hipMemcpyAsync(word_device,word,400* sizeof(char), hipMemcpyHostToDevice);
 // hipMemcpyAsync(word,word_device,400* sizeof(char), hipMemcpyDeviceToHost);
  int tid=threadIdx.x;
  int return_value=0;
 
 if( *(word+tid) != '\0')
  {
    //printf("in word_tid conditions");
   
    
   
    if ((x_shared=  *(c_trie_shared )) == NULL){
          new_container_shared(c_trie_shared, *word, word+1);
         // printf("in x_device conditions");
	}

    
    if( is_it_a_trie_shared(x_shared,trie_pack_entry_capacity_shared,trie_pack_idx_shared,trie_pack_shared)!=1 )
    {
       c_trie_shared = (char **)x_shared;
      // printf("is_trie");
       
    }
    else
    {
      //printf("is not trie"); 
      word++;

      
      if( *word == '\0')
      {
        //*(uint32_t *)(x_shared+2) = *(uint32_t *)(x_shared+2) + 1;
         return_value=1;
        // printf("in the word");  
         
      }
     // printf("in word");
     
    //if( (add_to_bucket_no_search_shared(x_shared, (word-1), word, c_trie_shared)) )
     // {
      // x_shared = *(c_trie_shared +  *(word-1));

     /* if( r > BUCKET_SIZE_LIM )
        {
          burst_container(x, *(word-1), c_trie);
        }*/

        
      //}
     
    }

    word++;
  }

  
}

void init()
{
  char **c_trie=NULL;
  int i=0;
  
  /* allocate the array of pointers that will be used to point to the
   * blocks of memory that house the trie nodes.
   */
  trie_pack = (char **) calloc (trie_pack_capacity, sizeof(char *));
  trie_pack_idx=0;
  trie_counter=0;

  /* assign the first pointer in the trie_pack array to block of memory */ 
  *(trie_pack+trie_pack_idx) =(char*) calloc(trie_pack_entry_capacity*TRIE_SIZE, sizeof(char));
  
  /* allocate a new trie node and assign it as the root trie node */
  root_trie=new_trie();
  c_trie = (char **)root_trie;

  /* make sure its pointers are null */
  for(i=0; i<128; i++)*(c_trie+i)=NULL; 
    

  /* make sure you clear the string-exhaust flag in the trie node */
  *(c_trie+STRING_EXHAUST_TRIE)=0;
}
__global__
void copy_string_to_array(char *dev_stri,char *dev_stri2,uint32_t register len)
{
  int tid=threadIdx.x;

  /*if( len < 128 )
  {
    *dev_stri = (char) len;


  }

  else
  {
    *dev_stri     = (char) ( len >> 8) | 0x80;
    *(++dev_stri) = (char) ( len ) & 0xff;

  }
  dev_stri++;
  */

 // if(tid<len)
 // {
  // array_dev[tid] = query_start_dev[tid];
  // printf("%s",array);
 // }
 // *array_dev='\0';
 // printf("%s",array_dev);

//int tID1=blockIdx.x;
  if( len < 128 )
  {
    *dev_stri = (char) len;
  }
  /* if the length of the string is greater than 128 characters, then two bytes are required to
   * store the string
   */
  else
  {
    *dev_stri     = (char) ( len >> 8) | 0x80;
      *(++dev_stri) = (char) ( len ) & 0xff;
 }
  dev_stri++;


 // if(dev_stri2[tID1]!='\0')
 //  {
   //     *dev_stri++=*dev_stri2++;
      //  printf("%d\n",tID1);

  // }
 // dev_stri=dev_stri2;

  int i=0;
  //printf("WMA's%s\n",dev_stri2);
  /*while(*dev_stri2!='\0')
  {
   *dev_stri++=*dev_stri2++;
   //printf("THIS %c\n",*dev_stri);
    i++;
   // printf("ARRAY IS%c\n",*array);
  }
 //*dev_stri='\0';
 // dev_stri=dev_stri2;
 //dev_stri=&dev_stri[0];
 //printf("THIS:::%c\n",dev_stri[0]);
  *dev_stri='\0';*/
  if(*(dev_stri2+tid)!='\0')
   {
        *(dev_stri+tid)=*(dev_stri2+tid);
      //  i++;
   }
   *(dev_stri+tid+1)='\0';
__syncthreads();
}



/* add a string to a container, using the techniques I developed for the HAT-trie.
 * This method simply appends a length-encoded string to the end of a bucket.
 */
uint32_t add_to_bucket_no_search(char *bucket,  
         char path, 
		     char *query_start, 
		     char **c_trie)
{
  char *array, *array_start, *query;
  char *tmp=*(c_trie+path);
    
  char *consumed=0;
  uint32_t array_offset;
  uint32_t register len;
  uint32_t num=0;

  array = (char *)(bucket+BUCKET_OVERHEAD);
  consumed = (char *)(bucket+CONSUMED);
  
  array_start=array;
  query = query_start;
 
  /* set a flag to indicate that the bucket now stores a string */
  if(*consumed == 0) { *consumed = 1; goto insert; }
  
  /* scan the container until you reach the null (end-of-bucket) character */
  while( *array != '\0') 
  {
  
    if( ( len = (unsigned int) *array ) >= 128 )
    {
      len = (unsigned int) ( ( *array & 0x7f ) << 8 ) |  (unsigned int) ( *(++array) & 0xff );
    }
    array = (array+1) + len;
    ++num;
  }

  insert:

  /* get the length of the string to insert */
 // for(; *query != '\0'; query++);//changed
   
// len = query - query_start;//changed
   
   len=slen(query);

  /* get the size of the array */
  array_offset = array-array_start;

  /* resize the array to fit the new string */
  resize_container((char **)(c_trie+path), array_offset, ( len < 128 ) ? len+2 : len+3);
 
  /* reinitialize the array pointers, the point to the end of the array */
  array = (char *)( *(c_trie+path) + BUCKET_OVERHEAD);
  array_start=array;  
  array += array_offset;
  increment++;
  /* if the length of the string is less than 128 characters, then only a single byte is
   * needed to store its length
   */
  if(increment<1500){

  if( len < 128 )
  {
    *array = (char) len;

  }  
  /* if the length of the string is greater than 128 characters, then two bytes are required to
   * store the string 
   */
  else 
  {
    *array     = (char) ( len >> 8) | 0x80;
    *(++array) = (char) ( len ) & 0xff; 

  }
  array++;
/* thrust::host_vector<char> G(sizeof(array));
  for(int i=0;i<sizeof(array);i++)
  {
        G[i]=array[i];
  }
  thrust::device_vector<char> C = G;*/

  /* copy the string into the array */ 
 /* while( *query_start != '\0')
  {
    *array++ = *query_start++;
  }*/
 // hipProfilerStart();
 /* hipStream_t mystream1;
   hipStream_t mystream2;
   hipStream_t mystream3;
  hipStreamCreate(&mystream1);
  hipStreamCreate(&mystream2);
  hipStreamCreate(&mystream3);*/
 /* hipStreamCreateWithFlags(&mystream1,hipStreamNonBlocking);
   hipStreamCreateWithFlags(&mystream2,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&mystream3,hipStreamNonBlocking);*/
  
 /* hipMemcpyAsync(&array_dev[0],&array[0],300*sizeof(char), hipMemcpyHostToDevice);
  hipMemcpyAsync(&query_start_dev[0],&query_start[0],300*sizeof(char), hipMemcpyHostToDevice);
  copy_string_to_array<<<1,len,0,mystream1>>>(&array_dev[0],&query_start_dev[0],len);
  //hipDeviceSynchronize();
  hipMemcpyAsync(&array[0],&array_dev[0],300*sizeof(char), hipMemcpyDeviceToHost);*/
   while( *query_start != '\0')
  {
    *array++ = *query_start++;
  }
   *array='\0';
}
else{
  /*
  while( *query_start != '\0')
  {
    *array++ = *query_start++;
  }*/
  hipMemcpyAsync(&array_dev[0],&array[0],300*sizeof(char), hipMemcpyHostToDevice);
  hipMemcpyAsync(&query_start_dev[0],&query_start[0],300*sizeof(char), hipMemcpyHostToDevice);
  copy_string_to_array<<<1,len,0,mystream1>>>(&array_dev[0],&query_start_dev[0],len);
  //hipDeviceSynchronize();
  hipMemcpyAsync(&array[0],&array_dev[0],300*sizeof(char), hipMemcpyDeviceToHost);
}
//  hipProfilerStop();
  /* make sure the array is null terminated */
 // *array='\0';
  ++num;

  return num;    
}

/*__global__
void copy_string(char* dev_stri,char* dev_stri2,int len)
{
   
    
   

  int tID=blockIdx.x*blockDim.x+threadIdx.x;
  int i=0;
 if(*(dev_stri2+tID)!='\0')
   {
        *(dev_stri+tID)=*(dev_stri2+tID);
        len--;
        
   }
   *(dev_stri+tID+1)='\0';
  __syncthreads();


}*/

/* add a string with its length to a container, using the techniques I developed for the HAT-trie.
 * This method simply appends a length-encoded string to the end of a bucket.
 */
uint32_t add_to_bucket_no_search_with_len(char *bucket,  
         char path, 
		     char *query_start, 
		     char **c_trie, int query_len)
{
  char *array, *array_start;
  char *tmp=*(c_trie+path);
  
  uint32_t len;
  char *consumed=0;
  uint32_t array_offset;

  array    = (char *)(bucket+BUCKET_OVERHEAD);
  consumed = (char *)(bucket+CONSUMED);
  
  array_start=array;

  /* set a flag to indicate that the bucket now stores a string */
  if(*consumed == 0) { *consumed = 1; goto insert; }
   
  /* scan the container until you reach the null (end-of-bucket) character */
  //integer
 /* while( *array != '\0')
  {
    if( ( len = (unsigned int) *array ) >= 128 )
    {
      len = (unsigned int) ( ( *array & 0x7f ) << 8 ) | (unsigned int) ( *(++array) & 0xff );
    }
    array = (array+1) + len;
  }*/
  
  insert:
  
  /* get the length of the string to insert */
  len = query_len;
  
  /* get the size of the array */
  array_offset = array-array_start;
   
  /* resize the array to fit the new string */
  resize_container((char **)(c_trie+path), array_offset, ( len < 128 ) ? len+2 : len+3);
   
  /* reinitialize the array pointers, the point to the end of the array */
  array = (char *)( *(c_trie+path) + BUCKET_OVERHEAD);
  array_start=array;  
  array += array_offset;
  
  /* if the length of the string is less than 128 characters, then only a single byte is
   * needed to store its length
   */
  if( len < 128 )
  {
    *array = (char) len;
  }
  /* if the length of the string is greater than 128 characters, then two bytes are required to
   * store the string 
   */
  else 
  {
    *array     = (char) ( len >> 8) | 0x80;
    *(++array) = (char) ( len ) & 0xff; 
  }
  array++;
  copy_increment++;
/*  thrust::host_vector<char> H(200);
  thrust::host_vector<char> F(200);
  
  for(int i=0; i<sizeof(array); i++)
  {
        H[i]=array[i];
  }
   for(int i=0;i<sizeof(query_start);i++)
  {
        F[i]=query_start[i];
  }*/
/*  thrust::device_vector<char> D = H;
  thrust::device_vector<char> G = F;
  dev_string1 = thrust::raw_pointer_cast(&D[0]); 
  dev_string2 = thrust::raw_pointer_cast(&G[0]);  
  copy_string<<<1,len>>>(dev_string1,dev_string2,len);*/
 // thrust::copy(d_b.begin(), d_b.end(), h_a.begin()); 
if(copy_increment<500){
  while( len!=0 )
  {
    *array++ = *query_start++;
    len--;
  }
  *array = '\0';
}
 else{
 // hipProfilerStart(); 
  hipMemcpyAsync(&dev_string1[0],&array[0],200*sizeof(char), hipMemcpyHostToDevice);
  hipMemcpyAsync(&dev_string2[0],&query_start[0],200*sizeof(char), hipMemcpyHostToDevice);
 copy_string<<<1,len>>>(&dev_string1[0],&dev_string2[0],len);//changing
  hipMemcpyAsync(&array[0],&dev_string1[0],200*sizeof(char), hipMemcpyDeviceToHost);
}
 // hipProfilerStop();
  /* make sure the array is null terminated */
  //*array = '\0';
  return 1;    
}

/* allocate a new container */
int new_container(char **c_trie, char path, char *word)
{
  char *x;
  
  /* allocate space for the container */

  x=(char*)malloc(BUCKET_OVERHEAD);
  if (x==NULL) fatal (MEMORY_EXHAUSTED);

  /* make sure the string-exhaust flag is cleared, and the
   * bytes used to store the pointer to the head of the list is
   * null.
   */
  *(x+CONSUMED)=0;
  *(uint32_t *)(x+STRING_EXHAUST_CONTAINER)=0;

   /* assign the parent pointer to the new container */
  *(c_trie + path)=x;
  
  if( *word == '\0')
  {
    *(uint32_t *)(x+STRING_EXHAUST_CONTAINER)=1;
  }
  else
  {
    add_to_bucket_no_search(x, path, word, c_trie); 
  }
  return 1;
}




int search(char *word)
{
  return 0;
}


/* insert a string into the copy based burst sort algorithm (i.e., burst trie) */
int insert(char *word)
{
 
  char **c_trie=  (char **) root_trie;
  char *x; 
  int r=0;
  
  
 // char *word_device;
  //hipMalloc((void **)&word_device, sizeof(word));
 // hipMemcpyAsync(word_device,word,400* sizeof(char), hipMemcpyHostToDevice);
 // hipMemcpyAsync(word,word_device,400* sizeof(char), hipMemcpyDeviceToHost);
 


 /* grab the leading character from the query string */
  while( *word != '\0')
  {
    /* if the pointer that maps to the leading character is null,
     * then create a new container to house the string, to complete
     * the insertion process
     */
    if ( (x = *(c_trie +  *word)) == NULL) 
      return new_container(c_trie, *word, word+1); 
         
    /* check whether the pointer that maps to the leading character 
     * leads to a trie node or to a container
     */
    if( is_it_a_trie(x) ) 
    {
       c_trie = (char **)x;
    }
    else
    {
      /* consume the lead character */
      word++;
      
      /* if the query string has been consumed entirely, then set
       * the string-exhaust flag within the current node to complete
       * the insertion 
       */
      if( *word == '\0') 
      { 
        *(uint32_t *)(x+STRING_EXHAUST_CONTAINER) = *(uint32_t *)(x+STRING_EXHAUST_CONTAINER) + 1;
        return 1;
      } 

      /* otherwise, a container is acquired.  Attempt to add the string
       * to the container.  If the function returns a non-null value,
       * then the insertion was a success. In this case, check to see
       * whether the container needs to be burst 
       */
      if( (r=add_to_bucket_no_search(x, *(word-1), word, c_trie)) )
      {
        x = *(c_trie +  *(word-1));

	 /* if the number of entries in the current container exceed the
         * container limit, then the container needs to be burst 
         */
        if( r > BUCKET_SIZE_LIM ) 
        {
	  burst_container(x, *(word-1), c_trie);
        }

        return 1;
      }
      return 0;
    }

    /* consume the current character and continue with the traversal */
    word++;
  }

  /* if the string was consumed prior to reaching a container, then 
   * set the string-exhaust flag within the current trie node to 
   * complete the insertion. 
   */
  *(uint64_t *)(c_trie+STRING_EXHAUST_TRIE) = *(uint64_t *)(c_trie+STRING_EXHAUST_TRIE) + 1;
  return 1;
} 

__global__ void adding(double *input,double*b,int num)
{
// printf("Entered");
/*static double sum;
 int tID = blockIdx.x * blockDim.x + threadIdx.x;
 int i=0;
 if (tID <= 2)
 {
 sum=sum+dev_a[i];
 i++;
 //printf("number %.2f\n",b[0]);
 __syncthreads();


  
 
// printf("TOTAL TIME AND TIDE IS %.2f\n",b[tID]);
 }
 b[0]=sum;*/
const int tid = threadIdx.x;

        auto int step_size = 1;
        int number_of_threads = blockDim.x;

        while (number_of_threads > 0)
        {
                if (tid < number_of_threads) // still alive?
                {
                        const auto int fst = tid * step_size * 2;
                        const auto int snd = fst + step_size;
                        input[fst] += input[snd];
                }

                step_size <<= 1;
                number_of_threads >>= 1;
        }
 //printf("TOTAL TIME AND TIDE IS %.6f\n",b[tID]);
}


int main(int argc, char **argv)
{
   char *to_insert=NULL, *to_search=NULL;
   int num_files=0;
   int i=0;
   int j=0;
   double mem=0;
   double insert_real_time=0.0, search_real_time=0.0;
  // hipProfilerStart();    
   hipMalloc((void **) &dev_string1,200*sizeof(char));
   hipMalloc((void **) &dev_string2,200*sizeof(char));
    hipMalloc((void **) &array_dev,500*sizeof(char));
   hipMalloc((void **) &query_start_dev,500*sizeof(char));
    hipMalloc((void **)&word_device, 500*sizeof(char));
   hipMalloc((void **)&trie_pack_device, trie_pack_capacity_device*sizeof(char*));
  hipMalloc((void**)&(*(trie_pack_device+trie_pack_idx_device)),trie_pack_entry_capacity_device*TRIE_SIZE_device*sizeof(char)); 
  hipMalloc((void**)&(*(trie_pack_device+trie_pack_idx_device)),(trie_pack_entry_capacity_device*TRIE_SIZE_device*sizeof(char)));
 // hipStreamCreate(&mystream1);
 // hipStreamCreate(&mystream2);
 //   hipMalloc((void **) &dest_device,500*sizeof(uint32_t));
 //   hipMalloc((void **) &src_device,500*sizeof(uint32_t));
  // hipMalloc((void**)&bucket_device, 200* sizeof(char*));
  // uint32_t *dest_device;
 // uint32_t *src_device;
  

 
   /* get the container limit */
   BUCKET_SIZE_LIM = atoi(argv[1]);

   /* make sure the user supplied a valid bucket size */
   if (BUCKET_SIZE_LIM < 64 || BUCKET_SIZE_LIM > 512)
   {
     puts("Keep bucket size between 128 and 256 strings, inclusive");
     exit(1);
   }

   /* since the bursting limit is actually a soft-limit, we need
    * to make room for some extra ptrs.
    */
   str_ptr = (ptr_struct *)calloc(BUCKET_SIZE_LIM*64, sizeof(ptr_struct *));
   path =(char*) calloc(524288, sizeof(char));

   /* get the number of files to insert */ 
   num_files = atoi(argv[2]);
 
   init();
   

   /* insert the files in sequence into the standard-chain burst trie and
    * accumulate the time required
    */
   
  /* for(i=0, j=3; i<num_files; i++, j++)
   {
     insert_real_time+=perform_insertion(argv[j]);
    
   }*/
   double e[num_files];
   int ce;
   for(i=0, j=3; i<num_files; i++, j++)
   {
     to_insert=argv[j];
     e[i]=perform_insertion(to_insert);
    // printf("%d\n",insert_real_time);
   // insert_real_time+=perform_insertion(to_insert);
   // printf("hurr %.2f\n",e[i]);
   }
  // printf("%d",insert_real_time);
   double a[num_files];
   double *dev_a;
    double *dev_b;
    double total;

   
   hipMalloc((void **) &dev_a, num_files*sizeof(double));
   hipMalloc((void **) &dev_b, num_files*sizeof(double));
  //hipStream_t *stream1 = (hipStream_t *) malloc(3 * sizeof(hipStream_t));
  //hipStreamCreate(stream1);
 //for(int i=0;i<3;i++){
 // hipStreamCreate(stream1);
  double result=0;

   hipMemcpy(dev_a,e, num_files*sizeof(double), hipMemcpyHostToDevice);
   adding<<<1,num_files>>>(dev_a,dev_b,num_files);

  hipMemcpy(&result,dev_a,sizeof(double), hipMemcpyDeviceToHost);
// hipProfilerStop(); 
  // printf("THIS IS ANS %.2f",a[0]);
   insert_real_time=result;
  hipFree(dev_a);
  hipFree(dev_b);
  
  /* thrust::device_vector<char*> H(num_files);
    for(int i=0;i<num_files;i++)
    {
        H[i]=argv[i];
    }*/

   uint64_t vsize=0;
   {
     pid_t mypid;
     FILE * statf;
     char fname[1024];
     uint64_t ret;
     uint64_t pid; 
     char commbuf[1024];
     char state;
     uint64_t ppid, pgrp, session, ttyd, tpgid;
     uint64_t flags, minflt, cminflt, majflt, cmajflt;
     uint64_t utime, stime, cutime, cstime, counter, priority;
     uint64_t timeout, itrealvalue;
     uint64_t starttime;
     uint64_t rss, rlim, startcode, endcode, startstack, kstkesp, ksteip;
     uint64_t signal, blocked, sigignore, sigcatch;
     uint64_t wchan;
     uint64_t size, resident, share, trs, drs, lrs, dt;
    
     mypid = getpid();
     snprintf(fname, 1024, "/proc/%u/stat", mypid);
     statf = fopen(fname, "r");
     ret = fscanf(statf, "%lu %s %c %lu %lu %lu %lu %lu %lu %lu %lu %lu %lu %lu %lu %lu %lu "
       "%lu %lu %lu %lu %lu %lu %lu %lu %lu %lu %lu %lu %lu %lu %lu %lu %lu %lu",
       &pid, commbuf, &state, &ppid, &pgrp, &session, &ttyd, &tpgid,
       &flags, &minflt, &cminflt, &majflt, &cmajflt, &utime, &stime,
       &cutime, &cstime, &counter, &priority, &timeout, &itrealvalue,
       &starttime, &vsize, &rss, &rlim, &startcode, &endcode, &startstack,
       &kstkesp, &ksteip, &signal, &blocked, &sigignore, &sigcatch,
       &wchan);
      
     if (ret != 35) {
        fprintf(stderr, "Failed to read all 35 fields, only %d decoded\n",
          ret);
     }
     fclose(statf);
   }
     /*         		dim3 dimBlock( blocksize, 1 );
        dim3 dimGrid( 1, 1 );
        destroy<<<dimGrid, dimBlock>>>();*/
	
  destroy();
   
   mem=((total_trie_pack_memory/(double)TO_MB) + ((double)bucket_mem/TO_MB));
   	
   fprintf(stderr, "Copybased burst sort %.2f %.2f %.2f %d %d --- A version of the burst-sort algorithm "
                   "implemented by Dr. Nikolas Askitis, Copyright @ 2016, askitisn@gmail.com ", vsize / (double) TO_MB, 
          mem, insert_real_time, get_inserted(), BUCKET_SIZE_LIM);
  printf("increment %d",copy_increment);
  
#ifdef PAGING
   fprintf(stderr, "%s\n", "Paging ");
#endif
 
#ifdef EXACT_FIT
   fprintf(stderr, "%s\n", "Exact-fit ");
#endif

   free(str_ptr);
   free(path);
  /* char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 
	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);
 
	printf("%s", a);
 
	hipMalloc( (void**)&ad, csize ); 
	hipMalloc( (void**)&bd, isize ); 
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
	
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
	hipFree( ad );
	hipFree( bd );
	
	printf("%s\n", a);*/
   // init_device<<<1,1>>>();

   return 0; 
}
//__global__
void burst_container(char *bucket, char path, char **c_trie)
{
    char *n_trie;

    /* allocate a new trie node as a parent */
    n_trie = new_trie();
    *(c_trie+path)=n_trie;
     
    c_trie = (char **) n_trie;  
    
    /* make sure you transfer the string-exhaust flag from the old container to the new trie node */
    uint64_t tmp=0; /* redundant step to ensure memory is zeroed */
    tmp =  (uint64_t) *(uint32_t *)(bucket+STRING_EXHAUST_CONTAINER);

    *(uint64_t *)(c_trie+STRING_EXHAUST_TRIE) = tmp;
    
    /* reset the string exhaust flag in the container */
    *(uint32_t *)(bucket+STRING_EXHAUST_CONTAINER)=0;

    /* split the container, passing the reference to the new trie node into the function */
    split_container(bucket, c_trie);
}

void split_container(char *bucket, char **c_trie)
{
  char *array = (char *)(bucket+BUCKET_OVERHEAD), *word_start;
  char *x;
  uint32_t len;

  /* scan the container until you reach the end-of-container (null) flag */
  while(*array != '\0')
  {
    /* get the length of the current string in the container */
    if( (len = (unsigned int) *array ) >= 128)
    {
      len = (unsigned int) ( ( *array & 0x7f ) << 8 ) | (unsigned int)  ( *(++array) & 0xff );
    }
  
    /* point to the first letter of the current string */
    array++;
    word_start = array;
   
    /* use the first letter to acquire a pointer in the parent trie */
    x = *(c_trie + *array);

    /* if the parent trie node pointer is null, then create a new container */  
    if (x == NULL)
    {
       /* allocate space for the container */
       x=(char*)malloc(BUCKET_OVERHEAD);
       if(x==NULL) fatal(MEMORY_EXHAUSTED);

       /* makes sure the string-exhaust and consumed flags are cleared and
        * assign the container to the parent trie
        */
       *(x+CONSUMED)=0;
       *(uint32_t *)(x+STRING_EXHAUST_CONTAINER)=0;
       *(c_trie + *array)=x;
    }   
    
    /* if after consuming the first character in the current string, you consume
     * the string, then set the string-exhaust flag in the current container
     */
    if( (len-1)==0 ) 
    {
      *(uint32_t *)(x+STRING_EXHAUST_CONTAINER) = *(uint32_t *)(x+STRING_EXHAUST_CONTAINER) + 1;
    }
    else
    {
      add_to_bucket_no_search_with_len(x, *array, array+1, c_trie, len-1); 
    }
    
    array = word_start  +  len;
  }
 
  /* you don't need the original bucket anymore */
  free(bucket);
}

/* run an in-order traversal of the burst trie to print out the strings
 * in ASCII-7 order, and also to accumulate the amount of memory 
 * allocated and to free the space allocated
 */
void in_order(char **c_trie, int local_depth, char *path)
{
  unsigned int i=0,j=0;
  char *x;
  
  if(local_depth > max_trie_depth)  max_trie_depth=local_depth;
  num_tries++;

  /* get the number of strings consumed by this trie */
  uint64_t num_consumed_trie = *(uint64_t *)(c_trie+STRING_EXHAUST_TRIE);

  for(j=0; j<num_consumed_trie; ++j)
  {
     printf("%s\n", path);         
  } 
  
  /* scan the trie node from left to right */
  for(i=MIN_RANGE; i<MAX_RANGE; i++)
  { 
    if ( (x = *(c_trie + i)) == NULL) 
    {
      continue;
    }

    path[local_depth-1]=(char)i;
    path[local_depth]='\0';
      
    if( is_it_a_trie(x) ) 
    {
      in_order( (char **)x, local_depth+1, path);
    }
    else
    {   
      char *x_start = x;
      char *tmp_str;
      unsigned int j=0;
      unsigned int k=0;
      unsigned int len=0;
      unsigned int num=0;
      unsigned int num_consumed_bucket=0;
      char *consumed=0;

      consumed = (char *)(x+CONSUMED);
      num_consumed_bucket=*(uint32_t *)(x+STRING_EXHAUST_CONTAINER);
      x=(char *)(x+BUCKET_OVERHEAD);

      for(j=0; j<num_consumed_bucket; ++j)
      { 
        printf("%s\n", path);         
      } 
 
      if(*consumed==1)
      {
         /* assign each string in the bucket to a pointer */
         while( *x != '\0')
         {
            if( ( len = (unsigned int) *x ) >= 128 )
            {
               len = (unsigned int) ( ( *x & 0x7f ) << 8 ) | (unsigned int) ( *(++x) & 0xff );
	    }
            ++x;      
            str_ptr[num].key=(uint8_t *)x; 
            str_ptr[num++].len=len;
            x=x+len;
         }

         /* sort the set of string pointers */
         tuned_qsort(str_ptr, num);

         /* iterate through the set of sorted string pointers to print out the strings */
         for(j=0; j<num; ++j)
         {
           tmp_str=(char *)str_ptr[j].key;
           len=str_ptr[j].len;

           /* we need to reconstruct the string before we print it, by storing
            * the path as the prefix. 
            */
           for(k=0; k<len; ++k)
           {
             *(path+local_depth+k)=*tmp_str;
             ++tmp_str;
           }
           *(path+local_depth+k)='\0';
           printf("%s\n", path);
         }
      }

#ifdef EXACT_FIT
      bucket_mem += ((x-x_start)+1); 
#else
      int temp= ((x-x_start)+1);

      if(temp<=_32_BYTES)
      {
        temp=_32_BYTES;
      }
      else 
      {
        if(temp <= _64_BYTES) 
        {
          temp = _64_BYTES;
        }
        else 
        {
          /* round up to the nearest 64-byte block */
          temp +=  _64_BYTES-(temp & (_64_BYTES -1 )); 
        }

        bucket_mem += temp; 
      }
#endif
      bucket_mem += ALLOC_OVERHEAD;
      num_buckets++;

      free(x_start);
      depth_accumulator+=local_depth;
     }
   }
}

/* free the memory allocated by the burst trie, including the trie nodes */

void destroy()
{
  int i=0;
  in_order((char **)root_trie, 1, path); 
  
  for(i=0; i<=trie_pack_idx; i++)  
  {
    total_trie_pack_memory += (((trie_pack_entry_capacity*TRIE_SIZE) + sizeof(char))+ALLOC_OVERHEAD);
    free ( *(trie_pack + i ) );
  }
  free(trie_pack);
}
